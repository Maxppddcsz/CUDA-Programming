#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>

#ifdef USE_DP
typedef double real;
#else
typedef float real;
#endif

const int NUM_REPEATS = 100;
const int N = 100000000;
const int M = sizeof(real) * N;
const int BLOCK_SIZE = 128;

void timing(real *h_x, real *d_x, const int method);

int main(void)
{
    real *h_x = (real *)malloc(M);
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1.23;
    }
    real *d_x;
    CHECK(hipMalloc(&d_x, M));

    printf("\nUsing global memory only:\n");
    timing(h_x, d_x, 0);
    printf("\nUsing static shared memory:\n");
    timing(h_x, d_x, 1);
    printf("\nUsing dynamic shared memory:\n");
    timing(h_x, d_x, 2);

    free(h_x);
    CHECK(hipFree(d_x));
    return 0;
}

void __global__ reduce_global(real *d_x, real *d_y)
{
    const int tid = threadIdx.x;
    real *x = d_x + blockDim.x * blockIdx.x;
    // real *x = &dx[blockDim.x*blockIdx.x];

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    // blockDim.x -> 线程块大小
    {
        if (tid < offset)
        {
            x[tid] += x[tid + offset];
        }
        __syncthreads(); // 保证和函数中语句的执行顺序与出现顺序一致，保证—个线程块中的所有线程（或者说所有线程束）在执行该语句后面的
                         // 语句之前都完全执行了该语句前面的语句
    }

    if (tid == 0)
    {
        d_y[blockIdx.x] = x[0];
    }
}

void __global__ reduce_shared(real *d_x, real *d_y)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    __shared__ real s_y[128]; // 将变量定义在共享内存变量中
    s_y[tid] = (n < N) ? d_x[n] : 0.0;
    __syncthreads(); // 在利用共享内存进行线程块之间的合作（通信）之前,都要进行同步
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {

        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        d_y[bid] = s_y[0];
    }
}

void __global__ reduce_dynamic(real *d_x, real *d_y)
{
    // 使用动态共享内存
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    extern __shared__ real s_y[]; // 动态共享定义时要添加关键词 extern；不能指定数组大小
    s_y[tid] = (n < N) ? d_x[n] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {

        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        d_y[bid] = s_y[0];
    }
}

real reduce(real *d_x, const int method)
{
    int grid_size = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    const int ymem = sizeof(real) * grid_size;
    const int smem = sizeof(real) * BLOCK_SIZE;
    real *d_y;
    CHECK(hipMalloc(&d_y, ymem));
    real *h_y = (real *)malloc(ymem);

    switch (method)
    {
    case 0:
        reduce_global<<<grid_size, BLOCK_SIZE>>>(d_x, d_y);
        break;
    case 1:
        reduce_shared<<<grid_size, BLOCK_SIZE>>>(d_x, d_y);
        break;
    case 2:
        reduce_dynamic<<<grid_size, BLOCK_SIZE, smem>>>(d_x, d_y);
        // 动态共享需要添加一个参数：核函数中每个线程块需要定义的动态共享内存的字节数
        break;
    default:
        printf("Error: wrong method\n");
        exit(1);
        break;
    }

    CHECK(hipMemcpy(h_y, d_y, ymem, hipMemcpyDeviceToHost));

    real result = 0.0;
    for (int n = 0; n < grid_size; ++n)
    {
        result += h_y[n];
    }

    free(h_y);
    CHECK(hipFree(d_y));
    return result;
}

void timing(real *h_x, real *d_x, const int method)
{
    real sum = 0;

    // for (int repeat = 0; repeat < NUM_REPEATS; ++repeat)
    // {
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start));
    hipEventQuery(start);

    sum = reduce(d_x, method);

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    float elapsed_time;
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Time = %g ms.\n", elapsed_time);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    //}

    printf("sum = %f.\n", sum);
}
